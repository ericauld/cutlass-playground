#include "hip/hip_runtime.h"
/*
Adapted from https://github.com/NVIDIA/cutlass/blob/main/examples/cute/tutorial/sgemm_1.cu
*/
#include <cstdlib>
#include <cstdio>
#include <cassert>

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>

#include <cute/tensor.hpp>

#include "cutlass/util/print_error.hpp"
#include "cutlass/util/GPU_Clock.hpp"
#include "cutlass/util/helper_cuda.hpp"

template <class ProblemShape, class CtaTiler,
          class TA, class AStride, class ASmemLayout, class AThreadLayout,
          class TB, class BStride, class BSmemLayout, class BThreadLayout,
          class TC, class CStride, class CSmemLayout, class CThreadLayout,
          class Alpha, class Beta>
__global__ static
__launch_bounds__(decltype(size(CThreadLayout{}))::value)
void
gemm_device(ProblemShape shape_MNK, CtaTiler cta_tiler,
            TA const* A, AStride dA, ASmemLayout sA_layout, AThreadLayout tA,
            TB const* B, BStride dB, BSmemLayout sB_layout, BThreadLayout tB,
            TC      * C, CStride dC, CSmemLayout          , CThreadLayout tC,
            Alpha alpha, Beta beta)
{
  using namespace cute;

  // Preconditions
  CUTE_STATIC_ASSERT_V(rank(shape_MNK) == Int<3>{});                   // (M, N, K)
  CUTE_STATIC_ASSERT_V(rank(cta_tiler) == Int<3>{});                   // (BLK_M, BLK_N, BLK_K)

  static_assert(is_static<AThreadLayout>::value);
  static_assert(is_static<BThreadLayout>::value);
  static_assert(is_static<CThreadLayout>::value);

  CUTE_STATIC_ASSERT_V(size(tA) == size(tB));                          // NumThreads
  CUTE_STATIC_ASSERT_V(size(tC) == size(tA));                          // NumThreads

  CUTE_STATIC_ASSERT_V(size<0>(cta_tiler) % size<0>(tA) == Int<0>{});  // BLK_M / THR_M
  CUTE_STATIC_ASSERT_V(size<2>(cta_tiler) % size<1>(tA) == Int<0>{});  // BLK_K / THR_K
  CUTE_STATIC_ASSERT_V(size<1>(cta_tiler) % size<0>(tB) == Int<0>{});  // BLK_N / THR_N
  CUTE_STATIC_ASSERT_V(size<2>(cta_tiler) % size<1>(tB) == Int<0>{});  // BLK_K / THR_K
  CUTE_STATIC_ASSERT_V(size<0>(cta_tiler) % size<0>(tC) == Int<0>{});  // BLK_M / THR_M
  CUTE_STATIC_ASSERT_V(size<1>(cta_tiler) % size<1>(tC) == Int<0>{});  // BLK_N / THR_N

  static_assert(is_static<ASmemLayout>::value);
  static_assert(is_static<BSmemLayout>::value);
  static_assert(is_static<CSmemLayout>::value);

  CUTE_STATIC_ASSERT_V(size<0>(ASmemLayout{}) == size<0>(cta_tiler));  // BLK_M
  CUTE_STATIC_ASSERT_V(size<1>(CSmemLayout{}) == size<0>(cta_tiler));  // BLK_M
  CUTE_STATIC_ASSERT_V(size<0>(BSmemLayout{}) == size<1>(cta_tiler));  // BLK_N
  CUTE_STATIC_ASSERT_V(size<1>(CSmemLayout{}) == size<1>(cta_tiler));  // BLK_N
  CUTE_STATIC_ASSERT_V(size<1>(ASmemLayout{}) == size<2>(cta_tiler));  // BLK_K
  CUTE_STATIC_ASSERT_V(size<1>(BSmemLayout{}) == size<2>(cta_tiler));  // BLK_K

  CUTE_STATIC_ASSERT_V(congruent(select<0,2>(shape_MNK), dA));         // dA strides for shape MK
  CUTE_STATIC_ASSERT_V(congruent(select<1,2>(shape_MNK), dB));         // dB strides for shape NK
  CUTE_STATIC_ASSERT_V(congruent(select<0,1>(shape_MNK), dC));         // dC strides for shape MN

  //
  // Full and Tiled Tensors
  //

  // Represent the full tensors
  Tensor mA = make_tensor(make_gmem_ptr(A), select<0,2>(shape_MNK), dA); // (M,K)
  Tensor mB = make_tensor(make_gmem_ptr(B), select<1,2>(shape_MNK), dB); // (N,K)
  Tensor mC = make_tensor(make_gmem_ptr(C), select<0,1>(shape_MNK), dC); // (M,N)

  // Get the appropriate blocks for this thread block
  auto cta_coord = make_coord(blockIdx.x, blockIdx.y, _);              // (m,n,k)
  Tensor gA = local_tile(mA, cta_tiler, cta_coord, Step<_1, X,_1>{});  // (BLK_M,BLK_K,k)
  Tensor gB = local_tile(mB, cta_tiler, cta_coord, Step< X,_1,_1>{});  // (BLK_N,BLK_K,k)
  Tensor gC = local_tile(mC, cta_tiler, cta_coord, Step<_1,_1, X>{});  // (BLK_M,BLK_N)

  // Shared memory buffers
  __shared__ TA smemA[cosize_v<ASmemLayout>];
  __shared__ TB smemB[cosize_v<BSmemLayout>];
  Tensor sA = make_tensor(make_smem_ptr(smemA), sA_layout);            // (BLK_M,BLK_K)
  Tensor sB = make_tensor(make_smem_ptr(smemB), sB_layout);            // (BLK_N,BLK_K)

  //
  // Partition the copying of A and B tiles across the threads
  //

  // TUTORIAL: Example of simple raked partitioning of ThreadLayouts tA|tB over data A|B tiles

  Tensor tAgA = local_partition(gA, tA, threadIdx.x);                  // (THR_M,THR_K,k)
  Tensor tAsA = local_partition(sA, tA, threadIdx.x);                  // (THR_M,THR_K)

  Tensor tBgB = local_partition(gB, tB, threadIdx.x);                  // (THR_N,THR_K,k)
  Tensor tBsB = local_partition(sB, tB, threadIdx.x);                  // (THR_N,THR_K)

  CUTE_STATIC_ASSERT_V(size<0>(tAgA) == size<0>(tAsA));                // THR_M
  CUTE_STATIC_ASSERT_V(size<1>(tAgA) == size<1>(tAsA));                // THR_K
  CUTE_STATIC_ASSERT_V(size<0>(tBgB) == size<0>(tBsB));                // THR_N
  CUTE_STATIC_ASSERT_V(size<1>(tBgB) == size<1>(tBsB));                // THR_K

  //
  // Define A/B partitioning and C accumulators
  //

  // TUTORIAL: Example of partitioning via projections of a ThreadLayout tC

  // Partition sA (M,K) by the rows of tC
  Tensor tCsA = local_partition(sA, tC, threadIdx.x, Step<_1, X>{});   // (THR_M,BLK_K)
  // Partition sB (N,K) by the cols of tC
  Tensor tCsB = local_partition(sB, tC, threadIdx.x, Step< X,_1>{});   // (THR_N,BLK_K)
  // Partition gC (M,N) by the tile of tC
  Tensor tCgC = local_partition(gC, tC, threadIdx.x, Step<_1,_1>{});   // (THR_M,THR_N)

  // Allocate the accumulators -- same shape/layout as the partitioned data
  Tensor tCrC = make_tensor_like(tCgC);                                // (THR_M,THR_N)

  CUTE_STATIC_ASSERT_V(size<0>(tCrC) == size<0>(tCgC));                // THR_M
  CUTE_STATIC_ASSERT_V(size<0>(tCrC) == size<0>(tCsA));                // THR_M
  CUTE_STATIC_ASSERT_V(size<1>(tCrC) == size<1>(tCgC));                // THR_N
  CUTE_STATIC_ASSERT_V(size<1>(tCrC) == size<0>(tCsB));                // THR_N
  CUTE_STATIC_ASSERT_V(size<1>(tCsA) == size<1>(tCsB));                // BLK_K

  // Clear the accumulators
  clear(tCrC);

// #if 0
  if(thread0()) {
    print("  mA : "); print(  mA); print("\n");
    print("  gA : "); print(  gA); print("\n");
    print("  sA : "); print(  sA); print("\n");
    print("tAgA : "); print(tAgA); print("\n");
    print("tAsA : "); print(tAsA); print("\n");
  }
// #endif

// #if 0
  if(thread0()) {
    print("  mB : "); print(  mB); print("\n");
    print("  gB : "); print(  gB); print("\n");
    print("  sB : "); print(  sB); print("\n");
    print("tBgB : "); print(tBgB); print("\n");
    print("tBsB : "); print(tBsB); print("\n");
  }
// #endif

// #if 0
  if(thread0()) {
    print("  mC : "); print(  mC); print("\n");
    print("  gC : "); print(  gC); print("\n");
    print("tCsA : "); print(tCsA); print("\n");
    print("tCsB : "); print(tCsB); print("\n");
    print("tCgC : "); print(tCgC); print("\n");
    print("tCrC : "); print(tCrC); print("\n");
  }
// #endif

/*
M = 5120
N = 5120
K = 4096

M = 5120
N = 5120
K = 4096

tAgA : (_4,_1,512):(_32,_0,40960)
tAsA : (_4,_1):(_32,_0)
tCsA : (_8,_8):(_16,_128)

tBgB : (_4,_1,512):(_32,_0,40960)
tBsB : (_4,_1):(_32,_0)
tCsB : (_8,_8):(_16,_128)

tCrC : (_8,_8):(_1,_8)
tCgC : (_8,_8):(_16,81920)

  mA : (5120,4096):(_1,5120)
  gA : (_128,_8,512):(_1,5120,40960)
  sA : (_128,_8):(_1,_128)

  mB : (5120,4096):(_1,5120)
  gB : (_128,_8,512):(_1,5120,40960)
  sB : (_128,_8):(_1,_128)

  mC : (5120,5120):(_1,5120)
  gC : (_128,_128):(_1,5120)
*/

#if 1

  // TUTORIAL: Example of a simple mainloop that read tiles of data into shared memory,
  //           and then computes on those tiles.
  //   copy(.) operates on the global and shared memory via the tA|tB partitioning
  //   gemm(.) operates on the shared and register memory via the tC partitioning

  auto K_TILE_MAX = size<2>(tAgA);

  for (int k_tile = 0; k_tile < K_TILE_MAX; ++k_tile)
  {
    // Copy gmem to smem with tA|tB thread-partitioned tensors
    copy(tAgA(_,_,k_tile), tAsA);      // A   (THR_M,THR_K) -> (THR_M,THR_K)
    copy(tBgB(_,_,k_tile), tBsB);      // B   (THR_N,THR_K) -> (THR_N,THR_K)

    // TUTORIAL: The above call to copy(tAgA(_,_,k_tile), tAsA) is equivalent to
    //   Tensor tAgAk = tAgA(_,_,k_tile);
    //   CUTE_UNROLL
    //   for (int i = 0; i < size(tAsA); ++i) {
    //     tAsA(i) = tAgAk(i);
    //   }

    cp_async_fence();        // Label the end of (potential) cp.async instructions
    cp_async_wait<0>();      // Sync on all (potential) cp.async instructions
    __syncthreads();         // Wait for all threads to write to smem

    // Compute gemm on tC thread-partitioned smem
    gemm(tCsA, tCsB, tCrC);            // (THR_M,THR_N) += (THR_M,BLK_K) * (THR_N,BLK_K)

    // TUTORIAL: The above call to gemm(tCsA, tCsB, tCrC) is equivalent to
    //   CUTE_UNROLL
    //   for (int k = 0; k < size<1>(tCsA); ++k) {
    //     CUTE_UNROLL
    //     for (int m = 0; m < size<0>(tCrC); ++m) {
    //       CUTE_UNROLL
    //       for (int n = 0; n < size<1>(tCrC); ++n) {
    //         tCrC(m,n) += tCsA(m,k) * tCsB(n,k);
    //       }
    //     }
    //   }

    __syncthreads();         // Wait for all threads to read from smem
  }

#endif

  //
  // Epilogue
  //

  axpby(alpha, tCrC, beta, tCgC);

  // TUTORIAL: The above call to axpby(alpha, tCrC, beta, tCgC) is equivalent to
  //   CUTE_UNROLL
  //   for (int i = 0; i < size(tCsA); ++i) {
  //     tCgC(i) = alpha * tCrC(i) + beta * tCgC(i);
  //   }
}

// Setup params for an NT GEMM
// Use m-major smem sA, n-major smem sB, and mn-major threads tA|tB
template <class TA, class TB, class TC,
          class Alpha, class Beta>
void
gemm_nt(int m, int n, int k,
        Alpha alpha,
        TA const* A, int ldA,
        TB const* B, int ldB,
        Beta beta,
        TC      * C, int ldC,
        hipStream_t stream = 0)
{
  using namespace cute;

  // Define shapes (dynamic)
  auto M = int(m);
  auto N = int(n);
  auto K = int(k);
  auto prob_shape = make_shape(M, N, K);                     // (M, N, K)

  // Define NT strides (mixed)
  auto dA = make_stride(Int<1>{}, ldA);                      // (dM, dK)
  auto dB = make_stride(Int<1>{}, ldB);                      // (dN, dK)
  auto dC = make_stride(Int<1>{}, ldC);                      // (dM, dN)

  // Define CTA tile sizes (static)
  auto bM = Int<128>{};
  auto bN = Int<128>{};
  auto bK = Int<  8>{};
  auto cta_tiler = make_shape(bM, bN, bK);                   // (BLK_M, BLK_N, BLK_K)

  // Define the smem layouts (static)
  auto sA = make_layout(make_shape(bM, bK));                 // (m,k) -> smem_idx; m-major
  auto sB = make_layout(make_shape(bN, bK));                 // (n,k) -> smem_idx; n-major
  auto sC = make_layout(make_shape(bM, bN));                 // (m,n) -> smem_idx; m-major

  // Define the thread layouts (static)
  auto tA = make_layout(make_shape(Int<32>{}, Int< 8>{}));   // (m,k) -> thr_idx
  auto tB = make_layout(make_shape(Int<32>{}, Int< 8>{}));   // (n,k) -> thr_idx
  auto tC = make_layout(make_shape(Int<16>{}, Int<16>{}));   // (m,n) -> thr_idx

  dim3 dimBlock(size(tC));
  dim3 dimGrid(size(ceil_div(M, bM)),
               size(ceil_div(N, bN)));
  gemm_device<<<dimGrid, dimBlock, 0, stream>>>
      (prob_shape, cta_tiler,
       A, dA, sA, tA,
       B, dB, sB, tB,
       C, dC, sC, tC,
       alpha, beta);
}

int main(int argc, char** argv)
{
  int m = 5120;
  int n = 5120;
  int k = 4096;
  
  char transA = 'N';
  char transB = 'T';

  using TA = float;
  using TB = float;
  using TC = float;
  using TI = float;

  TI alpha = 1.0;
  TI beta  = 0.0;

  std::cout << "M = " << m << std::endl;
  std::cout << "N = " << n << std::endl;
  std::cout << "K = " << k << std::endl;
  std::cout << "C = A^" << transA << " B^" << transB << std::endl;

  cute::device_init(0);

  thrust::host_vector<TA> h_A(m*k);
  thrust::host_vector<TB> h_B(n*k);
  thrust::host_vector<TC> h_C(m*n);

  for (int j = 0; j < m*k; ++j) h_A[j] = static_cast<TA>( 2*(rand() / double(RAND_MAX)) - 1 );
  for (int j = 0; j < n*k; ++j) h_B[j] = static_cast<TB>( 2*(rand() / double(RAND_MAX)) - 1 );
  for (int j = 0; j < m*n; ++j) h_C[j] = static_cast<TC>(-1);

  thrust::device_vector<TA> d_A = h_A;
  thrust::device_vector<TB> d_B = h_B;
  thrust::device_vector<TC> d_C = h_C;

  int ldA = m, ldB = n, ldC = m;

  d_C = h_C;
  gemm_nt(m, n, k, alpha, 
          d_A.data().get(), ldA, 
          d_B.data().get(), ldB, 
          beta, 
          d_C.data().get(), ldC, /*stream*/0);
  CUTE_CHECK_LAST();
  thrust::host_vector<TC> cute_result = d_C;

  CUTE_CHECK_LAST();
  return 0;
}
