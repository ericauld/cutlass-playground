#include "hip/hip_runtime.h"
#include <iostream>
#include <cassert>
#include <cmath>

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <cute/tensor.hpp>

void matrix_multiply_cpu(const cute::half_t* A, const cute::half_t* B, cute::half_t* C, int m, int n, int k) {
  for (int i = 0; i < m; ++i) {
    for (int j = 0; j < n; ++j) {
      cute::half_t sum = static_cast<cute::half_t>(0.0f);
      for (int p = 0; p < k; ++p) {
        sum += A[i * k + p] * B[j * k + p];
      }
      C[i * n + j] = sum;
    }
  }
}

bool areMatricesEqual(const cute::half_t* C1, const cute::half_t* C2, int m, int n, float tolerance = 1e-1) {
  for (int i = 0; i < m; ++i) {
    for (int j = 0; j < n; ++j) {
      if (std::fabs(static_cast<float>(C1[i * n + j]) - static_cast<float>(C2[i * n + j])) > tolerance) {
        return false;
      }
    }
  }
  return true;
}

template <class TiledMma>
__global__ static
void
f(cute::half_t const *A,
  cute::half_t const *B,
  cute::half_t       *C,
  int k1,
  TiledMma            my_mma) {
  using namespace cute;

  int k = k1 * 16;

  Tensor mA = make_tensor(make_gmem_ptr(A), make_layout(make_shape(_16{}, k), make_stride(k, _1{})));
  Tensor mB = make_tensor(make_gmem_ptr(B), make_layout(make_shape(_8{}, k), make_stride(k, _1{})));
  Tensor mC = make_tensor(make_gmem_ptr(C), make_layout(make_shape(_16{}, _8{}), make_stride(_8{}, _1{})));
  auto thrmma = my_mma.get_slice(threadIdx.x);

  // No need for gA, gB, or gC...only one CTA

  // Our single CTA has blockIdx.x = 0, blockIdx.y = 0
  auto cta_coord = make_coord(0, 0, _);
  auto cta_tiler = make_shape(_16{}, _8{}, _16{});
  Tensor gA = local_tile(mA, cta_tiler, cta_coord, Step<_1, X, _1>{});
  Tensor gB = local_tile(mB, cta_tiler, cta_coord, Step<X, _1, _1>{});

  Tensor tCgA = thrmma.partition_A(gA);
  Tensor tCgB = thrmma.partition_B(gB);
  Tensor tCmC = thrmma.partition_C(mC);

  auto rC = thrmma.make_fragment_C(tCmC);
  clear(rC);

  auto rA = thrmma.make_fragment_A(tCgA(_, _, _, 0));
  auto rB = thrmma.make_fragment_B(tCgB(_, _, _, 0));

#if 0
  if (thread0()) {
    print(my_mma);
    print("mA : "); print(mA); print("\n");
    print("mB : "); print(mB); print("\n");
    print("mC : "); print(mC); print("\n");
    print("gA : "); print(gA); print("\n");
    print("gB : "); print(gB); print("\n");
    print("tCgA : "); print(tCgA); print("\n");
    print("tCgB : "); print(tCgB); print("\n");
    print("tCmC : "); print(tCmC); print("\n");
    print("rA : "); print(rA); print("\n");
    print("rB : "); print(rB); print("\n");
    print("rC : "); print(rC); print("\n");
  }
#endif
#if 1
  for (int p1 = 0; p1 < k1; ++p1) {
    copy(tCgA(_, _, _, p1), rA);
    copy(tCgB(_, _, _, p1), rB);
    gemm(my_mma, rA, rB, rC);
  }
  copy(rC, tCmC);
#endif
  return;
}

void printMatrix(const cute::half_t* data, int m, int n) {
    std::cout << std::fixed << std::setprecision(4);
    for (int i = 0; i < m; ++i) {
        std::cout << "[ ";
        for (int j = 0; j < n; ++j) {
            std::cout << static_cast<float>(data[i * n + j]) << " ";
        }
        std::cout << "]" << std::endl;
    }
}

int main() {
  using namespace cute;

  int Xm = 16;
  int Xn = 8;
  int Xk = 16;

  int k1 = 7;

  int m = Xm;
  int n = Xn;
  int k = Xk * k1;

  using TA = half_t;

  thrust::host_vector<TA> h_A(m*k);
  thrust::host_vector<TA> h_B(n*k);
  thrust::host_vector<TA> h_C(m*n);

  for (int j = 0; j < m*k; ++j) h_A[j] = static_cast<TA>( 2*(rand() / double(RAND_MAX)) - 1 );
  for (int j = 0; j < n*k; ++j) h_B[j] = static_cast<TA>( 2*(rand() / double(RAND_MAX)) - 1 );
  for (int j = 0; j < m*n; ++j) h_C[j] = 0;

  thrust::device_vector<TA> d_A = h_A;
  thrust::device_vector<TA> d_B = h_B;
  thrust::device_vector<TA> d_C = h_C;

  using op = SM80_16x8x16_F16F16F16F16_TN;
  auto tiled_mma = make_tiled_mma(op{}, make_layout(make_shape(_1{}, _1{}, _1{}))); 

  dim3 dimGrid(1);
  dim3 dimBlock(32);
  
  f<<<dimGrid, dimBlock>>>(d_A.data().get(), d_B.data().get(), d_C.data().get(), k1, tiled_mma);

  thrust::host_vector<TA> cute_result = d_C;
#if 1
  matrix_multiply_cpu(h_A.data(), h_B.data(), h_C.data(), m, n, k);
#endif
#if 0
  print("h_A : "); printMatrix(h_A.data(), m, k); print("\n\n");
  print("h_B : "); printMatrix(h_B.data(), k, n); print("\n\n");
  print("h_C : "); printMatrix(h_C.data(), m, n); print("\n\n");
  print("cute_result : "); printMatrix(cute_result.data(), m, n); print("\n\n");
#endif
# if 1
  assert(areMatricesEqual(cute_result.data(), h_C.data(), m, n));
  std::cout << "Success!" << std::endl;
#endif
  return 0;
}
