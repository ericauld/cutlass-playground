#include "hip/hip_runtime.h"
#include <iostream>
#include <cassert>
#include <cmath>

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <cute/tensor.hpp>

void matrix_multiply_cpu(const cute::half_t* A, const cute::half_t* B, cute::half_t* C, int m, int n, int k) {
  for (int i = 0; i < m; ++i) {
    for (int j = 0; j < n; ++j) {
      cute::half_t sum = static_cast<cute::half_t>(0.0f);
      for (int p = 0; p < k; ++p) {
        sum += A[i * k + p] * B[j * k + p];
      }
      C[i * n + j] = sum;
    }
  }
}

bool areMatricesEqual(const cute::half_t* C1, const cute::half_t* C2, int m, int n, float tolerance = 1e-2) {
  for (int i = 0; i < m; ++i) {
    for (int j = 0; j < n; ++j) {
      if (std::fabs(static_cast<float>(C1[i * n + j]) - static_cast<float>(C2[i * n + j])) > tolerance) {
        return false;
      }
    }
  }
  return true;
}

template <class TiledMma>
__global__ static
void
f(cute::half_t const *A,
  cute::half_t const *B,
  cute::half_t       *C,
  int k1,
  TiledMma            my_mma) {
  using namespace cute;

  int k = k1 * 16;

  Tensor mA = make_tensor(make_gmem_ptr(A), make_layout(make_shape(_16{}, k), make_stride(k, _1{})));
  Tensor mB = make_tensor(make_gmem_ptr(B), make_layout(make_shape(_8{}, k), make_stride(k, _1{})));
  Tensor mC = make_tensor(make_gmem_ptr(C), make_layout(make_shape(_16{}, _8{}), make_stride(_8{}, _1{})));
  auto thrmma = my_mma.get_slice(threadIdx.x);

  // No need for gA, gB, or gC...only one CTA

  // Our single CTA has blockIdx.x = 0, blockIdx.y = 0
  auto cta_coord = make_coord(0, 0, _);
  auto cta_tiler = make_shape(_16{}, _8{}, _16{});
  Tensor gA = local_tile(mA, cta_tiler, cta_coord, Step<_1, X, _1>{});
  Tensor gB = local_tile(mB, cta_tiler, cta_coord, Step<X, _1, _1>{});

  Tensor tCgA = thrmma.partition_A(gA);
  Tensor tCgB = thrmma.partition_B(gB);
  Tensor tCmC = thrmma.partition_C(mC);

  auto rC = thrmma.partition_fragment_C(tCmC);
  clear(rC);

  auto rA = thrmma.partition_fragment_A(tCgA(_, _, _, 0));
  auto rB = thrmma.partition_fragment_B(tCgB(_, _, _, 0));

#if 1
  if (thread0()) {
    print(my_mma);
    print("mA : "); print(mA); print("\n");
    print("mB : "); print(mB); print("\n");
    print("mC : "); print(mC); print("\n");
    print("gA : "); print(gA); print("\n");
    print("gB : "); print(gB); print("\n");
    print("tCgA : "); print(tCgA); print("\n");
    print("tCgB : "); print(tCgB); print("\n");
    print("tCmC : "); print(tCmC); print("\n");
    print("rA : "); print(rA); print("\n");
    print("rB : "); print(rB); print("\n");
    print("rC : "); print(rC); print("\n");
  }
#endif
/*
TiledMMA
  ThrLayoutVMNK:  (_32,_1,_1,_1):(_1,_0,_0,_0)
  PermutationMNK: (_,_,_)
MMA_Atom
  ThrID:      _32:_1
  Shape_MNK:  (_16,_8,_16)
  LayoutA_TV: ((_4,_8),(_2,_2,_2)):((_32,_1),(_16,_8,_128))
  LayoutB_TV: ((_4,_8),(_2,_2)):((_16,_1),(_8,_64))
  LayoutC_TV: ((_4,_8),(_2,_2)):((_32,_1),(_16,_8))
mA : gmem_ptr[16b](0x7f5d5bc00000) o (_16,112):(112,_1)
mB : gmem_ptr[16b](0x7f5d5bc00e00) o (_8,112):(112,_1)
mC : gmem_ptr[16b](0x7f5d5bc01600) o (_16,_8):(_8,_1)
gA : gmem_ptr[16b](0x7f5d5bc00000) o (_16,_16,7):(112,_1,_16)
gB : gmem_ptr[16b](0x7f5d5bc00e00) o (_8,_16,7):(112,_1,_16)
tCgA : gmem_ptr[16b](0x7f5d5bc00000) o ((_2,_2,_2),_1,_1,7):((_1,896,_8),_0,_0,_16)
tCgB : gmem_ptr[16b](0x7f5d5bc00e00) o ((_2,_2),_1,_1,7):((_1,_8),_0,_0,_16)
tCmC : gmem_ptr[16b](0x7f5d5bc01600) o ((_2,_2),_1,_1):((_1,_64),_0,_0)
rA : ptr[16b](0x7f5d80fffcc0) o ((_2,_2,_2),_1,_1,_1):((_0,_1,_0),_0,_0,_0)
rB : ptr[16b](0x7f5d80fffcd0) o ((_2,_2),_1,_1,_1):((_0,_0),_0,_0,_0)
rC : ptr[16b](0x7f5d80fffcb0) o ((_2,_2),_1,_1,_1):((_1,_2),_0,_0,_0)
*/
#if 0
  for (int p1 = 0; p1 < k1; ++p1) {
    copy(tCgA(_, _, _, p1), rA);
    copy(tCgB(_, _, _, p1), rB);
    gemm(my_mma, rA, rB, rC);
  }
  copy(rC, tCmC);
#endif
  return;
}

void printMatrix(const cute::half_t* data, int m, int n) {
    std::cout << std::fixed << std::setprecision(4);
    for (int i = 0; i < m; ++i) {
        std::cout << "[ ";
        for (int j = 0; j < n; ++j) {
            std::cout << static_cast<float>(data[i * n + j]) << " ";
        }
        std::cout << "]" << std::endl;
    }
}

int main() {
  using namespace cute;

  int Xm = 16;
  int Xn = 8;
  int Xk = 16;

  int k1 = 7;

  int m = Xm;
  int n = Xn;
  int k = Xk * k1;

  using TA = half_t;

  thrust::host_vector<TA> h_A(m*k);
  thrust::host_vector<TA> h_B(n*k);
  thrust::host_vector<TA> h_C(m*n);

  for (int j = 0; j < m*k; ++j) h_A[j] = static_cast<TA>( 2*(rand() / double(RAND_MAX)) - 1 );
  for (int j = 0; j < n*k; ++j) h_B[j] = static_cast<TA>( 2*(rand() / double(RAND_MAX)) - 1 );
  for (int j = 0; j < m*n; ++j) h_C[j] = 0;

  thrust::device_vector<TA> d_A = h_A;
  thrust::device_vector<TA> d_B = h_B;
  thrust::device_vector<TA> d_C = h_C;

  using op = SM80_16x8x16_F16F16F16F16_TN;
  auto tiled_mma = make_tiled_mma(op{}, make_layout(make_shape(_1{}, _1{}, _1{}))); 

  dim3 dimGrid(1);
  dim3 dimBlock(32);
  
  f<<<dimGrid, dimBlock>>>(d_A.data().get(), d_B.data().get(), d_C.data().get(), k1, tiled_mma);

  thrust::host_vector<TA> cute_result = d_C;
#if 0
  matrix_multiply_cpu(h_A.data(), h_B.data(), h_C.data(), m, n, k);
#endif
#if 0
  print("h_A : "); printMatrix(h_A.data(), m, k); print("\n\n");
  print("h_B : "); printMatrix(h_B.data(), k, n); print("\n\n");
  print("h_C : "); printMatrix(h_C.data(), m, n); print("\n\n");
  print("cute_result : "); printMatrix(cute_result.data(), m, n); print("\n\n");
#endif
# if 0
  assert(areMatricesEqual(cute_result.data(), h_C.data(), m, n));
  std::cout << "Success!" << std::endl;
#endif
  return 0;
}
