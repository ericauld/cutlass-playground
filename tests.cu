#include "hip/hip_runtime.h"
#include <iostream>
#include <cassert>
#include <cmath>

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <cute/tensor.hpp>
#include "e.h"

void matrix_multiply_cpu(const cute::half_t* A, const cute::half_t* B, cute::half_t* C, int m, int n, int k) {
  for (int i = 0; i < m; ++i) {
    for (int j = 0; j < n; ++j) {
      cute::half_t sum = static_cast<cute::half_t>(0.0f);
      for (int p = 0; p < k; ++p) {
        sum += A[i * k + p] * B[j * k + p];
      }
      C[i * n + j] = sum;
    }
  }
}

bool areMatricesEqual(const cute::half_t* C1, const cute::half_t* C2, int m, int n, float tolerance = 1e-1f) {
  for (int i = 0; i < m; ++i) {
    for (int j = 0; j < n; ++j) {
      if (std::fabs(static_cast<float>(C1[i * n + j]) - static_cast<float>(C2[i * n + j])) > tolerance) {
        return false;
      }
    }
  }
  return true;
}

void printMatrix(const cute::half_t* data, int m, int n) {
    std::cout << std::fixed << std::setprecision(4);
    for (int i = 0; i < m; ++i) {
        std::cout << "[ ";
        for (int j = 0; j < n; ++j) {
            std::cout << static_cast<float>(data[i * n + j]) << " ";
        }
        std::cout << "]" << std::endl;
    }
}

void simplest() {
  using namespace cute;

  int Xm = 16;
  int Xn = 8;
  int Xk = 16;
  int Tm = 1;
  int Tn = 1;
  int Tk = 1;
  int m = Xm * Tm;
  int n = Xn * Tn;
  int k = Xk * Tk;

  using TA = half_t;

  thrust::host_vector<TA> h_A(m*k);
  thrust::host_vector<TA> h_B(k*n);
  thrust::host_vector<TA> h_C(m*n);

  for (int j = 0; j < m*k; ++j) h_A[j] = static_cast<TA>( 2*(rand() / double(RAND_MAX)) - 1 );
  for (int j = 0; j < n*k; ++j) h_B[j] = static_cast<TA>( 2*(rand() / double(RAND_MAX)) - 1 );
  for (int j = 0; j < m*n; ++j) h_C[j] = 0;

  thrust::device_vector<TA> d_A = h_A;
  thrust::device_vector<TA> d_B = h_B;
  thrust::device_vector<TA> d_C = h_C;

  using op = SM80_16x8x16_F16F16F16F16_TN;
  auto tiled_mma = make_tiled_mma(op{}, make_layout(make_shape(_1{}, _1{}, _1{}))); 

  dim3 dimGrid(1, 1);
  dim3 dimBlock(32);
  
  auto shape = make_shape(m, n, k);
  auto dA = make_stride(k, _1{});
  auto dB = make_stride(k, _1{});
  auto dC = make_stride(n, _1{});
  f<<<dimGrid, dimBlock>>>(d_A.data().get(), d_B.data().get(), d_C.data().get(), m, n, tiled_mma);

  thrust::host_vector<TA> cute_result = d_C;
#if 1
  matrix_multiply_cpu(h_A.data(), h_B.data(), h_C.data(), m, n, k);
#endif
#if 0
  print("h_A : "); printMatrix(h_A.data(), m, k); print("\n\n");
  print("h_B : "); printMatrix(h_B.data(), k, n); print("\n\n");
  print("h_C : "); printMatrix(h_C.data(), m, n); print("\n\n");
  print("cute_result : "); printMatrix(cute_result.data(), m, n); print("\n\n");
#endif
#if 1
  assert(areMatricesEqual(cute_result.data(), h_C.data(), m, n));
  std::cerr << "Simplest succeeded" << std::endl;
#endif
  return;
}

void second_simplest() {
  using namespace cute;

  int Xm = 16;
  int Xn = 8;
  int Xk = 16;
  int Tm = 4;
  int Tn = 5;
  int Tk = 1;
  int m = Xm * Tm;
  int n = Xn * Tn;
  int k = Xk * Tk;

  using TA = half_t;

  thrust::host_vector<TA> h_A(m*k);
  thrust::host_vector<TA> h_B(k*n);
  thrust::host_vector<TA> h_C(m*n);

  for (int j = 0; j < m*k; ++j) h_A[j] = static_cast<TA>( 2*(rand() / double(RAND_MAX)) - 1 );
  for (int j = 0; j < n*k; ++j) h_B[j] = static_cast<TA>( 2*(rand() / double(RAND_MAX)) - 1 );
  for (int j = 0; j < m*n; ++j) h_C[j] = 0;

  thrust::device_vector<TA> d_A = h_A;
  thrust::device_vector<TA> d_B = h_B;
  thrust::device_vector<TA> d_C = h_C;

  using op = SM80_16x8x16_F16F16F16F16_TN;
  auto tiled_mma = make_tiled_mma(op{}, make_layout(make_shape(_1{}, _1{}, _1{}))); 

  dim3 dimGrid(1, 1);
  dim3 dimBlock(32);
  
  auto shape = make_shape(m, n, k);
  auto dA = make_stride(k, _1{});
  auto dB = make_stride(k, _1{});
  auto dC = make_stride(n, _1{});
  f<<<dimGrid, dimBlock>>>(d_A.data().get(), d_B.data().get(), d_C.data().get(), m, n, tiled_mma);

  thrust::host_vector<TA> cute_result = d_C;
#if 1
  matrix_multiply_cpu(h_A.data(), h_B.data(), h_C.data(), m, n, k);
#endif
#if 0
  print("h_A : "); printMatrix(h_A.data(), m, k); print("\n\n");
  print("h_B : "); printMatrix(h_B.data(), k, n); print("\n\n");
  print("h_C : "); printMatrix(h_C.data(), m, n); print("\n\n");
  print("cute_result : "); printMatrix(cute_result.data(), m, n); print("\n\n");
#endif
#if 1
  assert(areMatricesEqual(cute_result.data(), h_C.data(), m, n));
  std::cerr << "Second simplest succeeded" << std::endl;
#endif
  return;
}

int main() {
  using namespace cute;

  simplest();
  second_simplest();

  int Xm = 16;
  int Xn = 8;
  int Xk = 16;

  int k1 = 7;

  int m = Xm;
  int n = Xn;
  int k = Xk * k1;

  using TA = half_t;

  thrust::host_vector<TA> h_A(m*k);
  thrust::host_vector<TA> h_B(n*k);
  thrust::host_vector<TA> h_C(m*n);

  for (int j = 0; j < m*k; ++j) h_A[j] = static_cast<TA>( 2*(rand() / double(RAND_MAX)) - 1 );
  for (int j = 0; j < n*k; ++j) h_B[j] = static_cast<TA>( 2*(rand() / double(RAND_MAX)) - 1 );
  for (int j = 0; j < m*n; ++j) h_C[j] = 0;

  thrust::device_vector<TA> d_A = h_A;
  thrust::device_vector<TA> d_B = h_B;
  thrust::device_vector<TA> d_C = h_C;

  using op = SM80_16x8x16_F16F16F16F16_TN;
  auto tiled_mma = make_tiled_mma(op{}, make_layout(make_shape(_1{}, _1{}, _1{}))); 

  dim3 dimGrid(1);
  dim3 dimBlock(32);
  
  f_local<<<dimGrid, dimBlock>>>(d_A.data().get(), d_B.data().get(), d_C.data().get(), k1, tiled_mma);

  thrust::host_vector<TA> cute_result = d_C;
#if 1
  matrix_multiply_cpu(h_A.data(), h_B.data(), h_C.data(), m, n, k);
#endif
#if 0
  print("h_A : "); printMatrix(h_A.data(), m, k); print("\n\n");
  print("h_B : "); printMatrix(h_B.data(), k, n); print("\n\n");
  print("h_C : "); printMatrix(h_C.data(), m, n); print("\n\n");
  print("cute_result : "); printMatrix(cute_result.data(), m, n); print("\n\n");
#endif
#if 1
  assert(areMatricesEqual(cute_result.data(), h_C.data(), m, n));
#endif
#if 1
  std::cerr << "Main succeeded" << std::endl;
#endif
  return 0;
}
