#include "hip/hip_runtime.h"
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <cute/tensor.hpp>

#include "cutlass/util/print_error.hpp"

#include <iostream>
#include <cassert>
#include <cmath>

// Simple CPU implementation of matrix multiplication, assuming A is transposed
void matrix_multiply_cpu(const cute::half_t* A, const cute::half_t* B, cute::half_t* C, int m, int n, int k) {
  for (int i = 0; i < m; ++i) {
    for (int j = 0; j < n; ++j) {
      cute::half_t sum = static_cast<cute::half_t>(0.0f);
      for (int p = 0; p < k; ++p) {
        // A is transposed, access it as A[p + i*k]
        sum += A[p + i * k] * B[p + j * k];
      }
      C[i * n + j] = sum;
    }
  }
}

bool areMatricesEqual(const cute::half_t* C1, const cute::half_t* C2, int m, int n, float tolerance = 1e-3) {
  for (int i = 0; i < m; ++i) {
    for (int j = 0; j < n; ++j) {
      if (std::fabs(static_cast<float>(C1[i * n + j]) - static_cast<float>(C2[i * n + j])) > tolerance) {
        return false;
      }
    }
  }
  return true;
}

template <class TiledMma>
__global__ static
void
f(cute::half_t const *A,
  cute::half_t const *B,
  cute::half_t       *C,
  TiledMma            my_mma) {
  using namespace cute;

  Tensor mA = make_tensor(make_gmem_ptr(A), make_layout(make_shape(_16{}, _16{}), make_stride(_16{}, _1{})));
  Tensor mB = make_tensor(make_gmem_ptr(B), make_shape(_8{}, _16{}));
  Tensor mC = make_tensor(make_gmem_ptr(C), make_shape(_16{}, _8{})); 
  auto thrmma = my_mma.get_slice(threadIdx.x);

  auto rC = thrmma.partition_fragment_C(mC);
  clear(rC);
  auto rA = thrmma.partition_fragment_A(mA);
  auto rB = thrmma.partition_fragment_B(mB);
  auto tCmC = thrmma.partition_C(mC);
  auto tCmA = thrmma.partition_A(mA);
  auto tCmB = thrmma.partition_B(mB);

#if 0
  print("mA : "); print(mA); print("\n");
  print("mB : "); print(mB); print("\n");
  print("mC : "); print(mC); print("\n");
  print("rA : "); print(rA); print("\n");
  print("rB : "); print(rB); print("\n");
  print("rC : "); print(rC); print("\n");
  print("tCmA : "); print(tCmA); print("\n");
  print("tCmB : "); print(tCmB); print("\n");
  print("tCmC : "); print(tCmC); print("\n");
#endif
#if 1
  copy(tCmA, rA);
  copy(tCmB, rB);
  gemm(my_mma, rA, rB, rC);
  copy(rC, tCmC);
#endif
  return;
}

void printMatrix(const cute::half_t* data, int rows, int cols) {
    std::cout << std::fixed << std::setprecision(4);
    for (int i = 0; i < rows; ++i) {
        std::cout << "[ ";
        for (int j = 0; j < cols; ++j) {
            std::cout << static_cast<float>(data[i * cols + j]) << " ";
        }
        std::cout << "]" << std::endl;
    }
}

int main() {
  using namespace cute;

  int m = 16;
  int n = 8;
  int k = 16;

  using TA = half_t;

  thrust::host_vector<TA> h_A(m*k);
  thrust::host_vector<TA> h_B(k*n);
  thrust::host_vector<TA> h_C(m*n);
  thrust::host_vector<TA> h_C_ref(m*n);

  for (int j = 0; j < m*k; ++j) h_A[j] = static_cast<TA>( 2*(rand() / double(RAND_MAX)) - 1 );
  for (int j = 0; j < n*k; ++j) h_B[j] = static_cast<TA>( 2*(rand() / double(RAND_MAX)) - 1 );
  for (int j = 0; j < m*n; ++j) h_C[j] = 0;

  thrust::device_vector<TA> d_A = h_A;
  thrust::device_vector<TA> d_B = h_B;
  thrust::device_vector<TA> d_C = h_C;

  matrix_multiply_cpu(h_A.data(), h_B.data(), h_C_ref.data(), m, n, k);

  using op = SM80_16x8x16_F16F16F16F16_TN;
  auto tiled_mma = make_tiled_mma(op{}, make_layout(make_shape(_1{}, _1{}, _1{}))); 

  dim3 dimGrid(1);
  dim3 dimBlock(32);
  f<<<dimGrid, dimBlock>>>(d_A.data().get(), d_B.data().get(), d_C.data().get(), tiled_mma);
  thrust::copy(d_C.begin(), d_C.end(), h_C.begin());
#if 1
  print("h_A : "); printMatrix(h_A.data(), m, k); print("\n\n");
  print("h_B : "); printMatrix(h_B.data(), k, n); print("\n\n");
  print("h_C : "); printMatrix(h_C.data(), m, n); print("\n\n");
  print("h_C_ref : "); printMatrix(h_C_ref.data(), m, n); print("\n\n");
#endif
  assert(areMatricesEqual(h_C.data(), h_C_ref.data(), m, n));

  std::cout << "Success!" << std::endl;
  return 0;
}
