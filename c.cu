#include "hip/hip_runtime.h"
#include <iostream>
#include <cassert>
#include <cmath>

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <cute/tensor.hpp>

void matrix_multiply_cpu(const cute::half_t* A, const cute::half_t* B, cute::half_t* C, int m, int n, int k) {
  for (int i = 0; i < m; ++i) {
    for (int j = 0; j < n; ++j) {
      cute::half_t sum = static_cast<cute::half_t>(0.0f);
      for (int p = 0; p < k; ++p) {
        sum += A[i * k + p] * B[j * k + p];
      }
      C[i * n + j] = sum;
    }
  }
}

bool areMatricesEqual(const cute::half_t* C1, const cute::half_t* C2, int m, int n, float tolerance = 1e-2f) {
  for (int i = 0; i < m; ++i) {
    for (int j = 0; j < n; ++j) {
      if (std::fabs(static_cast<float>(C1[i * n + j]) - static_cast<float>(C2[i * n + j])) > tolerance) {
        return false;
      }
    }
  }
  return true;
}

template <class TiledMma>
__global__ static
void
f(cute::half_t const *A,
  cute::half_t const *B,
  cute::half_t       *C,
  int m, int n,
  TiledMma            my_mma) {
  using namespace cute;

  Tensor mA = make_tensor(make_gmem_ptr(A), make_layout(make_shape(m, _16{}), make_stride(_16{}, _1{})));
  Tensor mB = make_tensor(make_gmem_ptr(B), make_layout(make_shape(n, _16{}), make_stride(_16{}, _1{})));
  Tensor mC = make_tensor(make_gmem_ptr(C), make_layout(make_shape(m, n), make_stride(n, _1{})));

  auto thrmma = my_mma.get_slice(threadIdx.x);

  auto cta_tiler = make_shape(_16{}, _8{}, _16{});
  auto cta_coord = make_coord(blockIdx.x, blockIdx.y, 0);
  Tensor gA = local_tile(mA, cta_tiler, cta_coord, Step<_1, X, _1>{});
  Tensor gB = local_tile(mB, cta_tiler, cta_coord, Step<X, _1, _1>{});
  Tensor gC = local_tile(mC, cta_tiler, cta_coord, Step<_1, _1, X>{});

  auto tCgA = thrmma.partition_A(gA);
  auto tCgB = thrmma.partition_B(gB);
  auto tCgC = thrmma.partition_C(gC);

  auto rC = thrmma.make_fragment_C(tCgC);
  clear(rC);
  auto rA = thrmma.make_fragment_A(tCgA);
  auto rB = thrmma.make_fragment_B(tCgB);

  copy(tCgA, rA);
  copy(tCgB, rB);
  gemm(my_mma, rA, rB, rC);
  copy(rC, tCgC);
  return;
}

void printMatrix(const cute::half_t* data, int m, int n) {
    std::cout << std::fixed << std::setprecision(4);
    for (int i = 0; i < m; ++i) {
        std::cout << "[ ";
        for (int j = 0; j < n; ++j) {
            std::cout << static_cast<float>(data[i * n + j]) << " ";
        }
        std::cout << "]" << std::endl;
    }
}

int main() {
  using namespace cute;

  int Xm = 16;
  int Xn = 8;
  int Xk = 16;
  int m1 = 5;
  int n1 = 6;
  int m = Xm * m1;
  int n = Xn * n1;
  int k = Xk;

  using TA = half_t;

  thrust::host_vector<TA> h_A(m * k);
  thrust::host_vector<TA> h_B(n * k);
  thrust::host_vector<TA> h_C(m * n);

  for (int j = 0; j < m*k; ++j) h_A[j] = static_cast<TA>( 2*(rand() / double(RAND_MAX)) - 1 );
  for (int j = 0; j < n*k; ++j) h_B[j] = static_cast<TA>( 2*(rand() / double(RAND_MAX)) - 1 );
  for (int j = 0; j < m*n; ++j) h_C[j] = 0;

  thrust::device_vector<TA> d_A = h_A;
  thrust::device_vector<TA> d_B = h_B;
  thrust::device_vector<TA> d_C = h_C;

  using op = SM80_16x8x16_F16F16F16F16_TN;
  auto tiled_mma = make_tiled_mma(op{}, make_layout(make_shape(_1{}, _1{}, _1{}))); 

  dim3 dimGrid(m1, n1);
  dim3 dimBlock(32);
  
  f<<<dimGrid, dimBlock>>>(d_A.data().get(), d_B.data().get(), d_C.data().get(),
                           m, n, tiled_mma);

  thrust::host_vector<TA> cute_result = d_C;
#if 1
  matrix_multiply_cpu(h_A.data(), h_B.data(), h_C.data(), m, n, k);
#endif
#if 0
  print("h_A : "); printMatrix(h_A.data(), m, k); print("\n\n");
  print("h_B : "); printMatrix(h_B.data(), k, n); print("\n\n");
  print("h_C : "); printMatrix(h_C.data(), m, n); print("\n\n");
  print("cute_result : "); printMatrix(cute_result.data(), m, n); print("\n\n");
#endif
#if 1
  assert(areMatricesEqual(cute_result.data(), h_C.data(), m, n));
  std::cout << "Success!" << std::endl;
#endif
  return 0;
}
