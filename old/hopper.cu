#include "hip/hip_runtime.h"
#include <cstdlib>
#include <cstdio>
#include <cassert>

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>

#include <cute/tensor.hpp>

#include "cutlass/util/print_error.hpp"
#include "cutlass/util/GPU_Clock.hpp"
#include "cutlass/util/helper_cuda.hpp"

template <class TiledMma>
__global__ static
void
f(cute::half_t const *A,
  cute::half_t const *B,
  cute::half_t       *C,
  TiledMma            my_mma) {
  using namespace cute;

  Tensor mA = make_tensor(make_gmem_ptr(A), make_shape(_16{}, _8{}));
  Tensor mB = make_tensor(make_gmem_ptr(B), make_shape(_8{}, _8{}));
  Tensor mC = make_tensor(make_gmem_ptr(C), make_shape(_16{}, _8{})); 

  // There is no distinction between mC and gC here because there's only one
  // block

  __shared__ half_t smemA[8*16];
  __shared__ half_t smemB[8*8];
  Tensor sA = make_tensor(make_smem_ptr(smemA), make_shape(_16{}, _8{}));
  Tensor sB = make_tensor(make_smem_ptr(smemB), make_shape(_8{}, _8{}));
  
  copy(mA, sA);
  copy(mB, sB);
  
  auto thrmma = my_mma.get_slice(threadIdx.x);

  auto rC = thrmma.partition_fragment_C(mC);
  auto rA = thrmma.partition_fragment_A(sA);
  auto rB = thrmma.partition_fragment_B(sB);

#if 0
  if(thread0()) {
    print("my_mma: "); print(my_mma); print("\n");
    print("sA: "); print(sA); print("\n");
    print("my_mma.thrfrg_A(sA) : "); print(my_mma.thrfrg_A(sA)); print("\n");
  }
#endif

#if 1
  gemm(my_mma, rA, rB, rC);

  auto tCmC = thrmma.partition_C(mC);
  copy(rC, tCmC);
#endif

}

void cpu_matmul(const cute::half_t *A, const cute::half_t *B, cute::half_t *C, int M, int N, int K) {
  for (int i = 0; i < M; ++i) {
    for (int j = 0; j < N; ++j) {
      float sum = 0;
      for (int k = 0; k < K; ++k) {
        float a = A[i*K + k];
        float b = B[k*N + j];
        sum += a * b;
      }
      C[i*N + j] = sum;
    }
  }
}

int main() {
  using namespace cute;

  int M = 16;
  int N = 8;
  int K = 8;

  using TA = half_t;

  thrust::host_vector<TA> h_A(M*K);
  thrust::host_vector<TA> h_B(K*N);
  thrust::host_vector<TA> h_C(M*N);

  for (int j = 0; j < M*K; ++j) h_A[j] = static_cast<TA>( 2*(rand() / double(RAND_MAX)) - 1 );
  for (int j = 0; j < N*K; ++j) h_B[j] = static_cast<TA>( 2*(rand() / double(RAND_MAX)) - 1 );
  for (int j = 0; j < M*N; ++j) h_C[j] = 0;

  thrust::device_vector<TA> d_A = h_A;
  thrust::device_vector<TA> d_B = h_B;
  // Why am I copying a bunch of zeros from host to device?
  thrust::device_vector<TA> d_C = h_C;

  using op = SM80_16x8x8_F16F16F16F16_TN;
  auto tiled_mma = make_tiled_mma(op{}, make_layout(make_shape(_1{}, _1{}, _1{}))); 

  dim3 dimGrid(1);
  dim3 dimBlock(32);
  f<<<dimGrid, dimBlock>>>(d_A.data().get(), d_B.data().get(), d_C.data().get(), tiled_mma);

  thrust::host_vector<TA> h_C2 = d_C;

#if 1
  cpu_matmul(h_A.data(), h_B.data(), h_C.data(), M, N, K);

  for (int i = 0; i < M; ++i) {
    for (int j = 0; j < N; ++j) {
      assert(h_C[i*N + j] == h_C2[i*N + j]);
    }
  }
#endif

  return 0;
}